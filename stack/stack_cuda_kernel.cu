#include "hip/hip_runtime.h"
// MIT License

// Copyright (c) Microsoft Corporation.

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE

#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

#define NUM_BLOCK 512

template <class T> __host__ __device__ T floorDiv(T a, T b) {
  T res = a / b, rem = a % b;
  return res - (rem != 0 && ((rem < 0) != (b < 0)));
}
template <class T> __host__ __device__ T ceilDiv(T a, T b) {
  T res = a / b, rem = a % b;
  return res + (rem != 0 && ((rem < 0) == (b < 0)));
}
template <class T> __host__ __device__ T runtime_mod(T a, T b) {
  T m = a % b;
  if (m < 0) {
      // m += (b < 0) ? -b : b; // avoid this form: it is UB when b == INT_MIN
      m = (b < 0) ? m - b : m + b;
  }
  return m;
}

template <typename scalar_t>
__global__ void kernel0(const scalar_t* input1, const scalar_t* input2, scalar_t* output, const int32_t stride, const int32_t elements) {
   if (((int)threadIdx.x < (min(((-NUM_BLOCK * (int)blockIdx.x) + (elements-1)), (NUM_BLOCK-1)) + 1))) {
     if ((runtime_mod(floorDiv<int32_t>((((int)blockIdx.x * NUM_BLOCK) + (int)threadIdx.x), stride), 2) == 0)) {
       output[(((int)blockIdx.x * NUM_BLOCK) + (int)threadIdx.x)] = input1[((floorDiv<int32_t>((((int)blockIdx.x * NUM_BLOCK) + (int)threadIdx.x), (stride * 2)) * stride) + runtime_mod((((int)blockIdx.x * NUM_BLOCK) + (int)threadIdx.x), stride))];
     }
     else {
       output[(((int)blockIdx.x * NUM_BLOCK) + (int)threadIdx.x)] = input2[((floorDiv<int32_t>((((int)blockIdx.x * NUM_BLOCK) + (int)threadIdx.x), (stride * 2)) * stride) + runtime_mod((((int)blockIdx.x * NUM_BLOCK) + (int)threadIdx.x), stride))];
     }
   }
 }


std::vector<torch::Tensor> stack_cuda_forward(
torch::Tensor input1, torch::Tensor input2, int stride)
{
    const int num_rows = input1.size(0) * 2;

    auto output = torch::zeros({num_rows}, torch::TensorOptions().device(torch::kCUDA));

    
    const dim3 block(NUM_BLOCK, 1, 1);
    const dim3 grid((num_rows - 1) / NUM_BLOCK + 1, 1, 1);

    AT_DISPATCH_FLOATING_TYPES(input1.type(), "stack_cuda_forward", ([&] {
        kernel0<scalar_t><<<grid, block>>>(
            input1.data<scalar_t>(),
            input2.data<scalar_t>(),
            output.data<scalar_t>(),
            stride,
            num_rows);
        }));
    
    return {output};
}

