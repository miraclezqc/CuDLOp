#include "hip/hip_runtime.h"
// MIT License

// Copyright (c) Microsoft Corporation.

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWAREfloorDiv

#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

#define NUM_BLOCK 512

template <class T> __host__ __device__ T floorDiv(T a, T b) {
    T res = a / b, rem = a % b;
    return res - (rem != 0 && ((rem < 0) != (b < 0)));
}
template <class T> __host__ __device__ T runtime_mod(T a, T b) {
    T m = a % b;
    if (m < 0) {
        // m += (b < 0) ? -b : b; // avoid this form: it is UB when b == INT_MIN
        m = (b < 0) ? m - b : m + b;
    }
    return m;
}

template <typename scalar_t>
__global__ void  kernel0(const scalar_t* input, scalar_t* output, const int elements, const int row, const int column, const int diag) {
    if (((int)threadIdx.x < (min(((-NUM_BLOCK * (int)blockIdx.x) + (elements-1)), (NUM_BLOCK-1)) + 1))) {
        int32_t r_idx;
        int32_t c_idx;
        c_idx = runtime_mod((((int)blockIdx.x * NUM_BLOCK) + (int)threadIdx.x), column);
        r_idx = runtime_mod(floorDiv<int32_t>((((int)blockIdx.x * NUM_BLOCK) + (int)threadIdx.x), column), row);
        if ((c_idx <= ((1 * diag) + r_idx))) {
          output[(((int)blockIdx.x * NUM_BLOCK) + (int)threadIdx.x)] = input[(((int)blockIdx.x * NUM_BLOCK) + (int)threadIdx.x)];
        }
        else {
          output[(((int)blockIdx.x * NUM_BLOCK) + (int)threadIdx.x)] = 0;
        }
      }
}



std::vector<torch::Tensor> tril_cuda_forward(
    torch::Tensor input, int row, int column, int diag)
{
    const int num_rows = input.size(0);

    auto output = torch::zeros({num_rows}, torch::TensorOptions().device(torch::kCUDA));

    
    const dim3 block(NUM_BLOCK, 1, 1);
    const dim3 grid((num_rows - 1) / NUM_BLOCK + 1, 1, 1);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "tril_cuda_forward", ([&] {
        kernel0<scalar_t><<<grid, block>>>(
            input.data<scalar_t>(),
            output.data<scalar_t>(),
            num_rows,
            row,
            column,
            diag);
        }));
    
    return {output};
}

